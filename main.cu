#include "hip/hip_runtime.h"
/*
 * EC527 Final Project
 * May 8, 2015
 * Gerardo Ravago - gerardo@gcr.me
 *
 * CUDA Based GPU Bitcoin Miner
 *
 * Special Thanks
 *  Brad Conte - Reference implementation of SHA-256
 */

#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <hip/hip_runtime.h>

#include "cuPrintf.cu"
#include "cuPrintf.cuh"
extern "C" {
#include "sha256.h"
#include "utils.h"
}
#include "sha256_unrolls.h"
#include "test.h"

// #define VERIFY_HASH		//Execute only 1 thread and verify manually
// #define ITERATE_BLOCKS	//Don't define BDIMX and create a 65535x1 Grid

/*
	Threads = BDIMX*GDIMX*GDIMY
	Thread Max = 2^32
	The most convenient way to form dimensions is to use a square grid of blocks
	GDIMX = sqrt(2^32/BDIMX)
*/

#define BDIMX 64   // MAX = 512
#define GDIMX 8192 // MAX = 65535 = 2^16-1
#define GDIMY GDIMX

__global__ void kernel_sha256d(SHA256_CTX *ctx, Nonce_result *nr);

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

#define CUDA_SAFE_CALL(ans)                         \
	{                                               \
		gpuAssert((ans), __FILE__, __LINE__);       \
	}

// Warning: This modifies the nonce value of data so do it last!
void compute_and_print_hash(unsigned char *data, unsigned int nonce)
{
	unsigned char hash[32];
	SHA256_CTX ctx;
	int i;

	*((unsigned long *)(data + 76)) = ENDIAN_SWAP_32(nonce);

	sha256_init(&ctx);
	sha256_update(&ctx, data, 80);
	sha256_final(&ctx, hash);
	sha256_init(&ctx);
	sha256_update(&ctx, hash, 32);
	sha256_final(&ctx, hash);

	printf("Data is: ");
	for (i = 0; i < 80; i++)
	{
		printf("%02X", data[i]);
	}
	printf("\n");
	printf("Hash is: ");
	for (i = 0; i < 8; i++)
	{
		printf("%.8x ", ENDIAN_SWAP_32(*(((unsigned int *)hash) + i)));
	}
	printf("\n");
}

int main(int argc, char **argv)
{
	unsigned char *data = test_block;

	// Initialize Cuda stuff
	cudaPrintfInit();
	dim3 DimGrid(GDIMX, GDIMY);
#ifndef ITERATE_BLOCKS
	dim3 DimBlock(BDIMX, 1);
#endif

	// Used to store a nonce if a block is mined
	Nonce_result h_nr;
	initialize_nonce_result(&h_nr);

	// Compute the shared portion of the SHA-256d calculation
	SHA256_CTX ctx;
	sha256_init(&ctx);
	sha256_update(&ctx, data, 80); // ctx.state contains a-h
	sha256_pad(&ctx);

	// Rearrange endianess of data to optimize device reads
	unsigned int *le_data = (unsigned int *)ctx.data;
	for (int i = 0, j = 0; i < 16; i++, j += 4)
	{
		le_data[i] = (ctx.data[j] << 24) | (ctx.data[j + 1] << 16) | (ctx.data[j + 2] << 8) | (ctx.data[j + 3]);
	}

	// Decodes and stores the difficulty in a 32-byte array for convenience
	unsigned int nBits = ENDIAN_SWAP_32(*((unsigned int *)(data + 72)));
	set_difficulty(ctx.difficulty, nBits);
	printf("nBits hex: %08X\n", nBits);
	printf("nBits int: %d\n", nBits);
	printf("Difficulty: %.8x\n", ctx.difficulty);

	// Allocate space on Global Memory
	SHA256_CTX *d_ctx;
	Nonce_result *d_nr;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_ctx, sizeof(SHA256_CTX)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_nr, sizeof(Nonce_result)));

	// Copy data to device
	CUDA_SAFE_CALL(hipMemcpy(d_ctx, &ctx, sizeof(SHA256_CTX), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_nr, &h_nr, sizeof(Nonce_result), hipMemcpyHostToDevice));

	float elapsed_gpu;
	long long int num_hashes;
#ifdef ITERATE_BLOCKS
	// Try different block sizes
	for (int i = 1; i <= 512; i++)
	{
		dim3 DimBlock(i, 1);
#endif
		// Start timers
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		// Launch Kernel
		kernel_sha256d<<<DimGrid, DimBlock>>>(d_ctx, d_nr);

		// Stop timers
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_gpu, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

#ifdef ITERATE_BLOCKS
		// Calculate results
		num_hashes = GDIMX * i;
		// block size, hashrate, hashes, execution time
		printf("%d, %.2f, %.0f, %.2f\n", i, num_hashes / (elapsed_gpu * 1e-3), num_hashes, elapsed_gpu);
	}
#endif

	CUDA_SAFE_CALL(hipMemcpy(&h_nr, d_nr, sizeof(Nonce_result), hipMemcpyDeviceToHost));

	// Cuda Printf output
	hipDeviceSynchronize();
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();

	// Free memory on device
	CUDA_SAFE_CALL(hipFree(d_ctx));
	CUDA_SAFE_CALL(hipFree(d_nr));

	// Output the results
	if (h_nr.nonce_found)
	{
		printf("Nonce found! %.8x  intNonce: %lld \n", h_nr.nonce, h_nr.nonce);
		compute_and_print_hash(data, h_nr.nonce);
	}
	else
	{
		printf("Nonce not found :(\n");
	}

	num_hashes = BDIMX;
	num_hashes *= GDIMX * GDIMY;
	printf("Tested %lld hashes\n", num_hashes);
	printf("GPU execution time: %f ms\n", elapsed_gpu);
	printf("Hashrate: %.2f H/s\n", num_hashes / (elapsed_gpu * 1e-3));

	return 0;
}

__constant__ uint32_t k[64] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

#define NONCE_VAL (gridDim.x * blockDim.x * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x)

__global__ void kernel_sha256d(SHA256_CTX *ctx, Nonce_result *nr)
{
	__shared__ int m[64];
	unsigned int hash[8];
	unsigned int a, b, c, d, e, f, g, h, i, t1, t2;
	unsigned int nonce = NONCE_VAL;

	unsigned int *le_data = (unsigned int *)ctx->data;
	for (int i = 0; i < 16; i++)
		m[i] = le_data[i];
	m[3] = nonce;
	for (int i = 16; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];

	SHA256_COMPRESS_8X

	m[0] = a + ctx->state[0];
	m[1] = b + ctx->state[1];
	m[2] = c + ctx->state[2];
	m[3] = d + ctx->state[3];
	m[4] = e + ctx->state[4];
	m[5] = f + ctx->state[5];
	m[6] = g + ctx->state[6];
	m[7] = h + ctx->state[7];
	m[8] = 0x80000000;
	for (int i = 9; i < 15; i++)
		m[i] = 0x00;
	m[15] = 0x00000100; // Write out l=256
	for (int i = 16; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	// Initialize the SHA-256 registers
	a = 0x6a09e667;
	b = 0xbb67ae85;
	c = 0x3c6ef372;
	d = 0xa54ff53a;
	e = 0x510e527f;
	f = 0x9b05688c;
	g = 0x1f83d9ab;
	h = 0x5be0cd19;

	SHA256_COMPRESS_1X

	hash[0] = ENDIAN_SWAP_32(a + 0x6a09e667);
	hash[1] = ENDIAN_SWAP_32(b + 0xbb67ae85);
	hash[2] = ENDIAN_SWAP_32(c + 0x3c6ef372);
	hash[3] = ENDIAN_SWAP_32(d + 0xa54ff53a);
	hash[4] = ENDIAN_SWAP_32(e + 0x510e527f);
	hash[5] = ENDIAN_SWAP_32(f + 0x9b05688c);
	hash[6] = ENDIAN_SWAP_32(g + 0x1f83d9ab);
	hash[7] = ENDIAN_SWAP_32(h + 0x5be0cd19);

	// Compare with difficulty
	bool found = true;
	for (int i = 0; i < 8; i++)
	{
		if (hash[i] < ctx->difficulty[i])
		{
			found = false;
			break;
		}
		else if (hash[i] > ctx->difficulty[i])
		{
			break;
		}
	}

	if (found)
	{
		atomicCAS(&nr->nonce_found, 0, 1);
		atomicExch(&nr->nonce, nonce);
	}
}
